
#include <hip/hip_runtime.h>
#include <stdio.h>
#ifndef THC_GENERIC_FILE
#define THC_GENERIC_FILE "generic/RotationInvariantEncoding.cu"
#else

int cuorn_(RIE_AlignFeature)(
    THCTensor *feature,
    THCudaByteTensor *mainDirection,
    THCTensor *aligned,
    const uint8 nOrientation)
{
    uint8 ndim = feature->nDimension;
    THArgCheck(ndim == 4, 1, "only supports batch mode.");

    const uint16 nBatch = feature->size[0];
    const uint16 nChannel = feature->size[1];
    const uint16 nFeature = nChannel / nOrientation;

    THCUNN_assertSameGPU(state, 3, feature, mainDirection, aligned);

    THCudaByteTensor_resize2d(state, mainDirection, nBatch, nFeature);
    THCTensor_(resizeAs)(state, aligned, feature);

    real *feature_data = THCTensor_(data)(state, feature);
    uint8 *mainDirection_data = THCudaByteTensor_data(state, mainDirection);
    real *aligned_data = THCTensor_(data)(state, aligned);

    const uint32 count = nBatch * nFeature;

    kernel_(AlignFeature)(
        THCState_getCurrentStream(state), 
        count, 
        feature_data, 
        nBatch, 
        nFeature, 
        nOrientation, 
        mainDirection_data, 
        aligned_data);
    THCudaCheck(cudaGetLastError());

    return 1;
}

int cuorn_(RIE_UnAlignFeature)(
    THCTensor *feature,
    THCudaByteTensor *mainDirection,
    THCTensor *aligned,
    const uint8 nOrientation)
{
    const uint16 nBatch = mainDirection->size[0];
    const uint16 nFeature = mainDirection->size[1];

    THCUNN_assertSameGPU(state, 3, feature, mainDirection, aligned);

    THCTensor_(resizeAs)(state, feature, aligned);

    real *feature_data = THCTensor_(data)(state, feature);
    uint8 *mainDirection_data = THCudaByteTensor_data(state, mainDirection);
    real *aligned_data = THCTensor_(data)(state, aligned);

    const uint32 count = nBatch * nFeature;

    kernel_(UnAlignFeature)(
        THCState_getCurrentStream(state), 
        count, 
        aligned_data, 
        mainDirection_data, 
        nBatch, 
        nFeature, 
        nOrientation, 
        feature_data);
    THCudaCheck(cudaGetLastError());
    
    return 1;
}

//////////////////////////////////////////////////////////////////////
int cuorn_(RIE_AlignFeature2d)(
    THCTensor *feature,
    THCudaByteTensor *mainDirection,
    THCTensor *aligned,
    const uint8 nOrientation)
{
    uint8 ndim = feature->nDimension;
    THArgCheck(ndim == 4, 1, "only supports batch mode.");

    const uint16 nBatch = feature->size[0];
    const uint16 nChannel = feature->size[1];
    const uint16 feature_h = feature->size[2];
    const uint16 feature_w = feature->size[3];
    const uint16 nFeature = nChannel / nOrientation;

    THCUNN_assertSameGPU(state, 3, feature, mainDirection, aligned);

    THCudaByteTensor_resize2d(state, mainDirection, nBatch, nFeature);
    THCTensor_(resizeAs)(state, aligned, feature);

    real *feature_data = THCTensor_(data)(state, feature);
    uint8 *mainDirection_data = THCudaByteTensor_data(state, mainDirection);
    real *aligned_data = THCTensor_(data)(state, aligned);

    const uint32 count = nBatch * nFeature;

    kernel_(AlignFeature2d)(
        THCState_getCurrentStream(state), 
        count, // count = nBatch * nFeature
        feature_data, // THCTensor_(data)(state, feature) feature: input
        nBatch, // nBatch = feature->size[0]
        nFeature, // nFeature = nChannel / nOrientation
        nOrientation, // 8
        feature_h,
        feature_w,
        mainDirection_data, //THCudaByteTensor_data(state, mainDirection)
        aligned_data); //THCTensor_(data)(state, aligned)
    THCudaCheck(cudaGetLastError());

    return 1;
}

int cuorn_(RIE_UnAlignFeature2d)(
    THCTensor *feature,
    THCudaByteTensor *mainDirection,
    THCTensor *aligned,
    const uint8 nOrientation)
{
    const uint16 nBatch = aligned->size[0];// 128
    const uint16 nChannel = aligned->size[1];// 640
    const uint16 nFeature = nChannel / nOrientation;// 80
    const uint16 feature_h = aligned->size[2];
    const uint16 feature_w = aligned->size[3];   
    THCUNN_assertSameGPU(state, 3, feature, mainDirection, aligned);

    THCTensor_(resizeAs)(state, feature, aligned);
    // printf("2");

    real *feature_data = THCTensor_(data)(state, feature);
    uint8 *mainDirection_data = THCudaByteTensor_data(state, mainDirection);
    real *aligned_data = THCTensor_(data)(state, aligned);

    const uint32 count = nBatch * nFeature;
    // printf("3");

    kernel_(UnAlignFeature2d)(
        THCState_getCurrentStream(state), 
        count, 
        aligned_data, 
        mainDirection_data, 
        nBatch, 
        nFeature, 
        nOrientation,
        feature_h,
        feature_w, 
        feature_data);
    THCudaCheck(cudaGetLastError());
    
    return 1;
}
////////////////////////////////////////////
#endif